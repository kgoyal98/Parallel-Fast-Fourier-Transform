#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <math.h>
#include <thrust/complex.h>
#include <sys/time.h>
#include <cassert>

using namespace std;

__constant__ const int block_1 = 16;
__constant__ const int block_2 = 8;


__global__
void fft_16(thrust::complex<double> *x, int l){
	const int n = block_1;
	int blidx = blockIdx.x, blidy = blockIdx.y;
	int k = blidx*l+blidy;
	int step = l/n;
	thrust::complex<double> temp[2*n];
	int pout[n], pin[n];
	// cout<<"fft-"<<n<<"\n";
	for(int i=0; i<n; i++){
		pout[i]=0;
		pin[i]=1;
		temp[pout[i]*n + i] = x[k+i*step];
		// cout<<k+i*step<<" ";
	}
	// cout<<endl;
	int thid1;
	for(int thid=0; thid<n; thid++){
		thid1 = 0;
		int b = __log2f(n+1);
		for(int i=0; i<b;i++){
		  if(thid & (1<<i))
		    thid1 |= (1<<(b-1-i));
		}
		pout[thid] = 1 - pout[thid];
		pin[thid] = 1 - pin[thid];
		temp[pout[thid]*n + thid] = temp[pin[thid]*n + thid1];
	}

	for(int i=1; i<n; i*=2){
		for(int thid=0; thid<n; thid++){
			pout[thid] = 1 - pout[thid];
			pin[thid] = 1 - pin[thid];
			thid1 = thid ^ i;
			thrust::complex<double> factor(cos(-M_PI*thid/i), sin(-M_PI*thid/i));
			if(thid1 > thid){
				temp[pout[thid]*n + thid] = temp[pin[thid]*n + thid] + factor * temp[pin[thid]*n + thid1];
			}
			else{
				temp[pout[thid]*n + thid] = temp[pin[thid]*n + thid1] + factor * temp[pin[thid]*n + thid];
			}
		}
	}
	for(int i=0; i<n; i++){
		thrust::complex<double> factor = thrust::complex<double>(cos(-M_PI*2*i*blidy/l), sin(-M_PI*2*i*blidy/l));
		// cout<<temp[pout[i]*n + i]<<"* "<<"("<<i<<","<<blidy<<")\n";
		x[k+i*step] = factor * temp[pout[i]*n + i];
	}
	// cout<<"\n--------------------\n";
}

__global__
void fft_8(thrust::complex<double> *x, int l){
	const int n = block_2;
	int blidx = blockIdx.x, blidy = blockIdx.y;
	int k = blidx*l+blidy;
	int step = l/n;
	thrust::complex<double> temp[2*n];
	int pout[n], pin[n];
	// cout<<"fft-"<<n<<"\n";
	for(int i=0; i<n; i++){
		pout[i]=0;
		pin[i]=1;
		temp[pout[i]*n + i] = x[k+i*step];
		// cout<<k+i*step<<" ";
	}
	// cout<<endl;
	int thid1;
	for(int thid=0; thid<n; thid++){
		thid1 = 0;
		int b = __log2f(n+1);
		for(int i=0; i<b;i++){
		  if(thid & (1<<i))
		    thid1 |= (1<<(b-1-i));
		}
		pout[thid] = 1 - pout[thid];
		pin[thid] = 1 - pin[thid];
		temp[pout[thid]*n + thid] = temp[pin[thid]*n + thid1];
	}

	for(int i=1; i<n; i*=2){
		for(int thid=0; thid<n; thid++){
			pout[thid] = 1 - pout[thid];
			pin[thid] = 1 - pin[thid];
			thid1 = thid ^ i;
			thrust::complex<double> factor(cos(-M_PI*thid/i), sin(-M_PI*thid/i));
			if(thid1 > thid){
				temp[pout[thid]*n + thid] = temp[pin[thid]*n + thid] + factor * temp[pin[thid]*n + thid1];
			}
			else{
				temp[pout[thid]*n + thid] = temp[pin[thid]*n + thid1] + factor * temp[pin[thid]*n + thid];
			}
		}
	}
	for(int i=0; i<n; i++){
		thrust::complex<double> factor = thrust::complex<double>(cos(-M_PI*2*i*blidy/l), sin(-M_PI*2*i*blidy/l));
		// cout<<temp[pout[i]*n + i]<<"* "<<"("<<i<<","<<blidy<<")\n";
		x[k+i*step] = factor * temp[pout[i]*n + i];
	}
	// cout<<"\n--------------------\n";
}

__global__
void fft_last(thrust::complex<double> *x, thrust::complex<double> *y, int x1, int x2){
	// cout<<"blidx "<<blidx<<endl;
	const int n = block_1;
	int blidx = blockIdx.x;
	int l=block_1, blidy=0;
	int k = blidx*l+blidy;
	int step = l/n;
	thrust::complex<double> temp[2*n];
	int pout[n], pin[n];
	// cout<<"fft-last\n";
	for(int i=0; i<n; i++){
		pout[i]=0;
		pin[i]=1;
		// cout<<"x["<<k+i*step<<"]"<<" "<<x[k+i*step]<<endl;
		temp[pout[i]*n + i] = x[k+i*step];
		// cout<<k+i*step<<" ";
	}
	// cout<<endl;
	int thid1;
	for(int thid=0; thid<n; thid++){
		thid1 = 0;
		int b = __log2f(n+1);
		for(int i=0; i<b;i++){
		  if(thid & (1<<i))
		    thid1 |= (1<<(b-1-i));
		}
		pout[thid] = 1 - pout[thid];
		pin[thid] = 1 - pin[thid];
		temp[pout[thid]*n + thid] = temp[pin[thid]*n + thid1];
	}

	for(int i=1; i<n; i*=2){
		for(int thid=0; thid<n; thid++){
			pout[thid] = 1 - pout[thid];
			pin[thid] = 1 - pin[thid];
			thid1 = thid ^ i;
			thrust::complex<double> factor(cos(-M_PI*thid/i), sin(-M_PI*thid/i));
			if(thid1 > thid){
				temp[pout[thid]*n + thid] = temp[pin[thid]*n + thid] + factor * temp[pin[thid]*n + thid1];
			}
			else{
				temp[pout[thid]*n + thid] = temp[pin[thid]*n + thid1] + factor * temp[pin[thid]*n + thid];
			}
		}
	}

	for(int i=0; i<n; i++){
		int p = blidx;
		int j = i;
		int loc = j;
		for(int k=0; k<x2; k++){
			int t = p&(block_2-1);
			loc = loc*block_2+t;
			p = p/block_2;
		}
		for(int k=0; k<x1-1; k++){
			int t = p&(block_1-1);
			loc = loc*block_1+t;
			p = p/block_1;
		}
		// cout<<"("<<blidx<<","<<i<<") -> ("<<loc/block_1<<","<<loc%block_1<<")\n";
		y[loc] = temp[pout[i]*n + i];
	}
}

void checkError(){
	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) 
	  printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
	  printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}


int main(){
	int n;
	cin>>n;
	thrust::complex<double> *x, *y;
	hipMallocManaged(&x, n*sizeof(thrust::complex<double>));
	hipMallocManaged(&y, n*sizeof(thrust::complex<double>));
	for(int i=0; i<n; i++){
		int t,u; cin>>t>>u;
		x[i] = thrust::complex<double>(t, u);
	}
	int m = log2(n+1);
	clock_t cpu_startTime, cpu_endTime;
	double cpu_ElapseTime=0;
	cpu_startTime = clock();
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	int x1,x2, log_block_1=log2(block_1+1), log_block_2=log2(block_2+1);
	for(int i=0; i<log_block_1; i++){
		if((m-log_block_2*i)%log_block_1 == 0){
			x1 = (m-log_block_2*i)/log_block_1;
			x2=i;
		}
	}
	// cout<<x1<<" "<<x2<<endl;
	int l = n;
	for(int i=0; i<x1-1; i++){
		dim3 grid(n/l, l/block_1, 1);
		dim3 block(1,1,1);
		fft_16<<<grid, block>>>(x, l);
		// checkError();
		l/=block_1;
	}
	// print(x, n);
	for(int i=0; i<x2; i++){
		dim3 grid(n/l, l/block_2, 1);
		dim3 block(1,1,1);
		fft_8<<<grid, block>>>(x, l);
		// checkError();
		l/=block_2;
	}
	assert(l==block_1);
	// print(x, n);
	// cout<<"l "<<l<<endl;
	dim3 grid(n/l, 1, 1);
	dim3 block(1,1,1);
	fft_last<<<grid, block>>>(x, y, x1, x2);
	// checkError();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	// cout<<milliseconds;
	// cpu_endTime = clock();
	// cpu_ElapseTime = (cpu_endTime - cpu_startTime);
	// cout<<cpu_ElapseTime;
	for(int i=0; i<n; i++){
		// if(i%(n/block_1)==0) cout<<endl;
		cout<<y[i]<<"\n";
	}
}