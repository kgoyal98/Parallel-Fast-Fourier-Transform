#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <math.h>
#include <thrust/complex.h>
#include <sys/time.h>
#include <cassert>
#include <hipfft/hipfft.h>

using namespace std;

int main(){
	int n;cin>>n;
	
	hipfftComplex *data_host = (hipfftComplex*) malloc (sizeof (hipfftComplex)* n);
	hipfftComplex *data_back = (hipfftComplex*) malloc (sizeof (hipfftComplex)* n);
	for(int i=0; i<n; i++){
		cin>>data_host[i].x;
		cin>>data_host[i].y;
	}
	clock_t cpu_startTime, cpu_endTime;
	double cpu_ElapseTime=0;
	cpu_startTime = clock();
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	hipfftHandle plan;
	hipfftComplex *data1;
	hipMalloc ((void **) &data1, sizeof(hipfftComplex)*n);
	hipMemcpy(data1, data_host, n*sizeof(hipfftComplex), hipMemcpyHostToDevice);
		
	int batch=1;
	hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);
	hipfftExecC2C(plan, data1, data1, HIPFFT_FORWARD);
	//hipfftExecC2C(plan, data1, data1, HIPFFT_BACKWARD);
	// cudaSynchronize();
	// cpu_endTime = clock();
	// cpu_ElapseTime = (cpu_endTime - cpu_startTime);
	// cout<<cpu_ElapseTime<<" ";
	
	hipEventRecord(stop);
	hipMemcpy(data_back, data1, n*sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<milliseconds;
	hipfftDestroy(plan);
	for(int i=0; i<n; i++){
		// cout<<"("<<data_back[i].x<<","<<data_back[i].y<<")"<<endl;					
	}
}
