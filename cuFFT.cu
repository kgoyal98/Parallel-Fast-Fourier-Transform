#include <iostream>
#include <complex>
#include <math.h>
#include <thrust/complex.h>
#include <sys/time.h>
#include <cassert>
#include <hipfft/hipfft.h>

using namespace std;

int main(){
	int n;cin>>n;
	
	hipfftComplex *data_host = (hipfftComplex*) malloc (sizeof (hipfftComplex)* n);
	hipfftComplex *data_back = (hipfftComplex*) malloc (sizeof (hipfftComplex)* n);
	for(int i=0; i<n; i++){
		cin>>data_host[i].x;
		cin>>data_host[i].y;
	}
	hipfftHandle plan;
	hipfftComplex *data1;
	hipMalloc ((void **) &data1, sizeof(hipfftComplex)*n);
	hipMemcpy(data1, data_host, n*sizeof(hipfftComplex), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	int batch=1;
	hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);
	hipfftExecC2C(plan, data1, data1, HIPFFT_FORWARD);

	hipEventRecord(stop);

	hipMemcpy(data_back, data1, n*sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	cout<<milliseconds;
	
	hipfftDestroy(plan);
	// for(int i=0; i<n; i++){
		// cout<<"("<<data_back[i].x<<","<<data_back[i].y<<")"<<endl;					
	// }
}
